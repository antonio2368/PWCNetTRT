#include "hip/hip_runtime.h"
#include "LeakyReluLayerGpu.h"

__global__ void LeakyRelu( const int n, const float* in, float* out, float alpha )
{
    CUDA_KERNEL_LOOP(index, n)
    {
        out[ index ] = in[ index ] < 0.0f ? in[ index ] * alpha : in[ index ];
    }
}

hipError_t LeakyReluForward( const int count, const float* input, float* output, float alpha )
{
    LeakyRelu<<< cudaBlockNum( count ), CUDA_THREADS_NUM >>>( count, input, output, alpha );
}
